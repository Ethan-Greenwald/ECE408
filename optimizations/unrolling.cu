#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#define TILE_WIDTH 16

__global__ void conv_forward_kernel(float* __restrict__ output, const float* __restrict__ input, const float* __restrict__ mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    int W_size = ceil((float)Width_out/TILE_WIDTH);
    int H_size = ceil((float)Height_out/TILE_WIDTH);

    int map = blockIdx.x;
    int batch = blockIdx.y;
    int h = blockIdx.z / W_size * TILE_WIDTH + threadIdx.y;    //output height
    int w = blockIdx.z % W_size * TILE_WIDTH + threadIdx.x;                 //output width
    
    /* Each thread in the block calculates its output value */
    if(h < Height_out && w < Width_out){
        float acc = 0.0f;

        #pragma unroll
        for(int c = 0; c < Channel; c++){

            #pragma unroll
            for(int p = 0; p < K; p++){

                #pragma unroll
                for(int q = 0; q < K; q++)
                    if(!(h+p > Height || w + q > Width))    //bounds check
                        acc += in_4d(batch, c, h + p, w + q) * mask_4d(map, c, p, q);
            }
        }

        out_4d(batch, map, h, w) = acc;
    }

    #undef out_4d
    #undef in_4d
    #undef mask_4d
}
	
__global__ void mat_mul_conv(float* __restrict__ output, const float* __restrict__ input, const float* __restrict__ mask, const int Map_out, const int Channel, const int Height, const int Width, const int K) {
    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]//feature, channel, row, col

    __shared__ float maskTile[TILE_WIDTH][TILE_WIDTH];
    __shared__ float inputTile[TILE_WIDTH][TILE_WIDTH];

    /* General util variables */
    int batch = blockIdx.z;
    int tx = threadIdx.x, ty = threadIdx.y;
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int numACol = Channel * K * K;

    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    
    float result = 0.0;

    int numTiles = ceil((float)numACol/TILE_WIDTH);
        #pragma unroll
        for (int i = 0; i < numTiles; i++) {
            /* Unroll and load in mask/input tiles */
            int load_col = i * TILE_WIDTH + tx;
            int load_row = i * TILE_WIDTH + ty;

            int c = load_col / (K * K);           //channel = column / mask_size
            int h = (load_col % (K * K)) / K;     //row = (column % mask_size) / mask_size
            int w = (load_col % (K * K)) % K;     //col = (column % mask_size) % mask_size

            /* Load in mask tile value */
            if (load_col < numACol && row < Map_out) 
                maskTile[ty][tx] = mask_4d(row, c, h, w);////////
            else 
                maskTile[ty][tx] = 0.0;

            c = load_row / (K * K);
            h = col / Width_out;
            w = col % Width_out;
            
            int p = load_row % (K * K) / K;     //offset based on mask row
            int q = (load_row % (K * K)) % K;   //offset based on mask col

            /* Load in input tile value */
            if (load_row < numACol && col < Height_out*Width_out) 
                inputTile[ty][tx] = in_4d(batch, c, h+p, w+q);/////////////
            else 
                inputTile[ty][tx] = 0.0;
            __syncthreads();

            /* Calculate partial dot product */
            if((row < Map_out) && (col < Width_out*Height_out)){
                #pragma unroll
                for(int i = 0; i < TILE_WIDTH; i++)
                    result += maskTile[ty][i]*inputTile[i][tx];
            }
            __syncthreads();
        }

    /* Store output (with bounds check) */
    if ((row < Map_out) && (col < Width_out*Height_out))
        out_4d(batch, row, col / Width_out, col % Width_out) = result;//////////
    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    hipMalloc((void**)device_input_ptr, sizeof(float) * Height * Width * Channel * Batch);
    hipMalloc((void**)device_output_ptr, sizeof(float) * Batch * Map_out * (Height-K+1) * (Width-K+1));
    hipMalloc((void**)device_mask_ptr, sizeof(float) * Map_out * Channel * K * K);

    hipMemcpy(*device_input_ptr, host_input, sizeof(float) * Height * Width * Channel * Batch, hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, sizeof(float) * Map_out * Channel * K * K, hipMemcpyHostToDevice);
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Set the kernel dimensions and call the kernel
    // get_device_properties();
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    dim3 grid(Map_out, Batch, ceil((float)(Height)/TILE_WIDTH) * ceil((float)(Width)/TILE_WIDTH));
    dim3 mat_mul_grid(ceil((float)(Height_out * Width_out)/TILE_WIDTH), ceil((float)Map_out/TILE_WIDTH), Batch);
    dim3 block(TILE_WIDTH, TILE_WIDTH, 1);

    bool isFirstLayer = Map_out < 10;
    if(isFirstLayer)
        conv_forward_kernel<<<grid, block>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);
    else
        mat_mul_conv<<<mat_mul_grid, block>>>(device_output, device_input, device_mask, Map_out, Channel, Height, Width, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    hipMemcpy(host_output, device_output, sizeof(float) * Batch * Map_out * (Height-K+1) * (Width-K+1), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
